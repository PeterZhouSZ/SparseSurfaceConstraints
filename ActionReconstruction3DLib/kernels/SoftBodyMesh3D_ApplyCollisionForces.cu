#include "hip/hip_runtime.h"
#include "../SoftBodyMesh3D.h"
#include "../CommonKernels.h"
#include "../Utils3D.h"

namespace ar3d
{
	__global__ void MeshApplyCollisionForcesKernel(dim3 size,
		const real3* positions, const real3* displacements, const real3* velocities,
		real4 groundPlane, real groundStiffness, real softminAlpha, real timestep, real theta,
		real3* forces)
	{
		CUMAT_KERNEL_1D_LOOP(i, size)
			real3 pos = positions[i] + displacements[i];
			real3 vel = velocities[i];
			real4 normalDist = SoftBodySimulation3D::groundDistance(groundPlane, pos);
			real softmin = ar3d::utils::softmin(normalDist.w, softminAlpha);
			real distDt = SoftBodySimulation3D::groundDistanceDt(groundPlane, vel);
			real fCurrent = -groundStiffness * softmin; //current timestep
			real fDt = -groundStiffness * (ar3d::utils::softminDx(normalDist.w, softminAlpha) * distDt); //time derivative
			real fNext = fCurrent + timestep * fDt; //next timestep
			real f = theta * fNext + (1 - theta) * fCurrent; //average force magnitude
			forces[i] += f * make_real3(normalDist.x, normalDist.y, normalDist.z); //final force
		CUMAT_KERNEL_1D_LOOP_END
	}

	void SoftBodyMesh3D::applyCollisionForces(const Input& input, const Settings& settings, const State& state,
		Vector3X& bodyForces)
	{
		//For simplicity, I assume every node has the same boundary length, which is equal to one.
		//This means, no correction for tet size is applied.
		//This has to be controlled by the groundStiffness in the settings

		cuMat::Context& ctx = cuMat::Context::current();
		cuMat::KernelLaunchConfig cfg = ctx.createLaunchConfig1D(input.numFreeNodes_, MeshApplyCollisionForcesKernel);
		
		const real3* positions = input.referencePositions_.data();
		const real3* displacements = state.displacements_.data();
		const real3* velocities = state.velocities_.data();
		real3* forces = bodyForces.data();

		MeshApplyCollisionForcesKernel <<< cfg.block_count, cfg.thread_per_block, 0, ctx.stream() >>>(
			cfg.virtual_size, positions, displacements, velocities, 
			settings.groundPlane_, settings.groundStiffness_, settings.softmaxAlpha_, settings.timestep_, settings.newmarkTheta_,
			forces);
		CUMAT_CHECK_ERROR();
	}
}
